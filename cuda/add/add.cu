
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    int *d_a, *d_b, *d_c;  // device pointers
    int h_a = 2, h_b = 5, h_c;  // host variables

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // Copy host data to device
    hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum %d!\n", h_c);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
