#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 500000000
#define THREADS_PER_BLOCK 1

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
    {
        c[index] = a[index] + b[index];
    }
}

int main(void)
{
    int *h_a, *h_b, *h_c;    // Host vectors
    int *d_a, *d_b, *d_c;    // Device vectors
    int size = N * sizeof(int);
    hipEvent_t start, stop;
    float milliseconds = 0;

    // Allocate host memory
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++)
    {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Create CUDA events for timing
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start);

    // Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel
    int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    vectorAdd<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Time taken for GPU operation: %f ms\n", milliseconds);

    // Verify the result
    //for (int i = 0; i < N; i++)
    //{
    //    printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    //}

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
